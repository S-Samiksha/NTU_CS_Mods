#include "hip/hip_runtime.h"
#include <stdio.h>
__global__ void hello_gpu(void){

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    printf("Hello from GPU[%d]! \n", i);

}

int main(void){
printf("Hello, Would - from CPU!");
hello_gpu<<1,4>>();
hipDeviceSynchronize();
hello_gpu<<1,6>>();
cudaDeviceSychronize();
return 0;

}