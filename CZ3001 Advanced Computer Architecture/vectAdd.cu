#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void vector_add_cu(int *d_c, int *d_a, int *d_b, int n){ 
    int i = blockIdx.x * blockDim.x + threadIdx.x; 
    c[i] = d_a[i] + d_b[i]; 

}
int main(void){
    int N = 4, i=0;
    int a[N] = {22,13,16, 5};
    int b[N] = {5, 22, 17, 37};
    int c[N]; 
    int *d_a, *d_b, *d_c; 
    hipMalloc((void**)&d_a, sizeof(int)*N); 
    hipMalloc((void**)&d_b, sizeof(int)*N); 
    hipMalloc((void**)&d_c, sizeof(int)*N); 
    hipMemcpy(d_a, a, sizeof(int)*N, hipMemcpyHostToDevice); 
    hipMemcpy(d_b, b, sizeof(int)*N, hipMemcpyHostToDevice); 
    vector_add_cu<<<1,N>>>(d_c, d_a, d_b, N); // note: 1 thread
    hipDeviceSynchronize();
    hipMemcpy(c, d_c, sizeof(int)*N, hipMemcpyDeviceToHost); 
    printf("value of d_c: ");
    for (i=0; i<N;i++){
        printf(" %d ", d_c[i]);
    }
    printf("\n");
    hipFree(d_a); 
    hipFree(d_b);
    hipFree(d_c);
  
} 
