#include "hip/hip_runtime.h"
#include <stdio.h>
__global__ void dot_prod_cu(int *d_c, int *d_a, int *d_b){ 
    __shared__ int tmp[3]; 
    int i = threadIdx.x;
    tmp[i] = d_a[i] * d_b[i]; 
    __syncthreads();
    if (i==0){
        int sum = 0, j=0;
        for (int j = 0; j < 3; j++){
            sum = sum + tmp[j]; 
            *c = sum; 
        }
        
    } 
}

int main(void){
    int N = 4, i=0;
    int a[N] = {22,13,16, 5};
    int b[N] = {5, 22, 17, 37};
    int c[N]; 
    int *d_a, *d_b, *d_c; 
    hipMalloc((void**)&d_a, sizeof(int)*N); 
    hipMalloc((void**)&d_b, sizeof(int)*N); 
    hipMalloc((void**)&d_c, sizeof(int)); 
    hipMemcpy(d_a, a, sizeof(int)*N, hipMemcpyHostToDevice); 
    hipMemcpy(d_b, b, sizeof(int)*N, hipMemcpyHostToDevice); 
    dot_prod_cu<<<1,N>>>(d_c, d_a, d_b); // note: 1 thread
    hipDeviceSynchronize();
    hipMemcpy(c, d_c, sizeof(int), hipMemcpyDeviceToHost); 
    printf("value of d_c: %d", &d_c);
    printf("\n");
    hipFree(d_a); 
    hipFree(d_b);
    hipFree(d_c);
  
} 